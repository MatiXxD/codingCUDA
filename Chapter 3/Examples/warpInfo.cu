#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CHECK(call) {                                                          \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


int main(int argc, char *argv[]) {

    int iDev = 0;
    hipDeviceProp_t iProp;
    CHECK(hipGetDeviceProperties(&iProp, iDev));

    printf("Device %d: %s\n", iDev, iProp.name);
    printf("  Number of multiprocessors:                     %d\n",
           iProp.multiProcessorCount);
    printf("  Total amount of constant memory:               %4.2f KB\n",
           iProp.totalConstMem / 1024.0);
    printf("  Total amount of shared memory per block:       %4.2f KB\n",
           iProp.sharedMemPerBlock / 1024.0);
    printf("  Total number of registers available per block: %d\n",
           iProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           iProp.warpSize);
    printf("  Maximum number of threads per block:           %d\n",
           iProp.maxThreadsPerBlock);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           iProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of warps per multiprocessor:    %d\n",
           iProp.maxThreadsPerMultiProcessor / 32);

    return EXIT_SUCCESS;

}