#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CHECK(call) {                                                          \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


__global__ void unrolling8v1(int *g_idata, int *g_odata, unsigned int n){

    unsigned int tid = threadIdx.x;                                 
    unsigned int idx = threadIdx.x + 8 * blockIdx.x * blockDim.x;

    int* idata = g_idata + 8 * blockIdx.x * blockDim.x;                           // Get data for four blocks 

    if(idx + 7*blockDim.x < n){                                                   // Each tread add element from the neighboring data blocks
        g_idata[idx] += g_idata[idx + 1*blockDim.x];
        g_idata[idx] += g_idata[idx + 2*blockDim.x];
        g_idata[idx] += g_idata[idx + 3*blockDim.x];
        g_idata[idx] += g_idata[idx + 4*blockDim.x];
        g_idata[idx] += g_idata[idx + 5*blockDim.x];
        g_idata[idx] += g_idata[idx + 6*blockDim.x];
        g_idata[idx] += g_idata[idx + 7*blockDim.x];
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) 
            idata[tid] += idata[tid + stride];
        __syncthreads();
    }
    
    if (tid == 0) g_odata[blockIdx.x] = idata[0];

}

__global__ void unrolling8v2(int *g_idata, int *g_odata, unsigned int n){

    unsigned int tid = threadIdx.x;                                 
    unsigned int idx = threadIdx.x + 8 * blockIdx.x * blockDim.x;

    int* idata = g_idata + 8 * blockIdx.x * blockDim.x;                               // Get data for four blocks 

    int *ptr = g_idata + idx;           
    int tmp = 0;
    if(idx + 7*blockDim.x < n){
        for (int i = 0; i < 8; i++) {                                                 // Each tread add element from the neighboring data blocks     
            tmp += *ptr; 
            ptr += blockDim.x;
        }
        g_idata[idx] = tmp;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) 
            idata[tid] += idata[tid + stride];
        __syncthreads();
    }
    
    if (tid == 0) g_odata[blockIdx.x] = idata[0];

}

double seconds() {
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

int recursiveReduce(int *data, int const size) {

    if (size == 1) return data[0];

    int const stride = size / 2;
    for (int i = 0; i < stride; i++)
        data[i] += data[i + stride];

    return recursiveReduce(data, stride);

}


int main(int argc, char **argv) {

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);
    bool bResult = false;

    // initialization
    int size = 1<<24; // total number of elements to reduce
    printf(" with array size %d ", size);

    // execution configuration
    int blocksize = 512;                                                            // initial block size
    if(argc > 1) 
        blocksize = atoi(argv[1]);                                                  // block size from command line argument
    dim3 block (blocksize,1);
    dim3 grid ((size+block.x-1)/block.x,1);
    printf("grid %d block %d\n",grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x*sizeof(int));
    int *tmp = (int *) malloc(bytes);

    // initialize the array
    for (int i = 0; i < size; i++) 
        h_idata[i] = (int)(rand() & 0xFF);                                          // mask off high 2 bytes to force max number to 255
    memcpy (tmp, h_idata, bytes);
    size_t iStart,iElaps;
    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x*sizeof(int));

    // cpu reduction
    iStart = seconds ();
    int cpu_sum = recursiveReduce(tmp, size);
    iElaps = seconds () - iStart;
    printf("cpu reduce elapsed %d ms cpu_sum: %d\n",iElaps,cpu_sum);

    // gpu unrolling8v1 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds ();
    unrolling8v1<<<grid.x/8, block.x>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds () - iStart;
    hipMemcpy(h_odata, d_odata, (grid.x/8)*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i=0; i < grid.x/8; i++) gpu_sum += h_odata[i];
    printf("gpu unrolling8v1 elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n",
        iElaps,gpu_sum,grid.x,block.x);
    if(gpu_sum != cpu_sum) bResult = true;

    // gpu unrolling8v2
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds ();
    unrolling8v2<<<grid.x/8, block.x>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds () - iStart;
    hipMemcpy(h_odata, d_odata, (grid.x/8)*sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i=0; i < grid.x/8; i++) gpu_sum += h_odata[i];
    printf("gpu unrolling8v2 elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n",
        iElaps,gpu_sum,grid.x,block.x);
    if(gpu_sum != cpu_sum) bResult = true;

    /// free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    hipFree(d_idata);
    hipFree(d_odata);

    // reset device
    hipDeviceReset();

    if(bResult) puts("Arrays doesn't match.");

    return EXIT_SUCCESS;

}